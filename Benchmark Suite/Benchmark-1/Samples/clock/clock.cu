#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 6
#define INF 999

__global__ void dfs(int *d_graph, int *d_visited, int *d_over){
    int id = threadIdx.x;
    if(id < N){
        for(int i=0; i<N; i++){
            if(d_graph[id*N + i] == 1 && d_visited[i] == 0){
                d_over[0] = 1;
                d_visited[i] = 1;
                dfs<<<1,N>>>(d_graph, d_visited, d_over);
            }
        }
    }
}

int main(){
int h_graph[N*N] = {0, 1, 0, 0, 0, 0,
                    1, 0, 1, 0, 0, 0,
                    0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0};

    int h_visited[N] = {1, 0, 0, 0, 0, 0};

    int *d_graph, *d_visited, *d_over;

    hipMalloc((void**)&d_graph, sizeof(int)*N*N);
    hipMalloc((void**)&d_visited, sizeof(int)*N);
    hipMalloc((void**)&d_over, sizeof(int));

    hipMemcpy(d_graph, h_graph, sizeof(int)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_visited, h_visited, sizeof(int)*N, hipMemcpyHostToDevice);

    int h_over;
    do{
        h_over = 0;
        hipMemcpy(d_over, &h_over, sizeof(int), hipMemcpyHostToDevice);
        dfs<<<1,N>>>(d_graph, d_visited, d_over);
        hipMemcpy(&h_over, d_over, sizeof(int), hipMemcpyDeviceToHost);
    }while(h_over == 1);

    hipMemcpy(h_visited, d_visited, sizeof(int)*N, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        if(h_visited[i] == 1)
            printf("%d ", i);
    }

    hipFree(d_graph);
    hipFree(d_visited);
    hipFree(d_over);

    return 0;
}