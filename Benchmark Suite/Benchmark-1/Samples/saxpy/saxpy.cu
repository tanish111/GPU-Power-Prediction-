#include "hip/hip_runtime.h"
#include <stdio.h>

#include <stdlib.h>
#include <iostream>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include "GPUDevice.h"

using namespace std;

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char **argv) {

	printf("Saxpy\n");

	int numBlocks, numThreadsPerBlock, N;

	hipEvent_t start,stop;
	float kerneltime;

	numBlocks = G_S;
	numThreadsPerBlock = B_S;
	N = numBlocks * numThreadsPerBlock;

	 cout << "\n" << "No. of blocks: " << numBlocks << "\nThreads per block: " << numThreadsPerBlock << "\nTotal no. of threads = " << N << "\n\n";

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// kernel
	// for (i = 0; i < 1000; i++)
	float *d_x, *d_y;
	float x[N], y[N];
	for (int i = 0; i < N; i++) {
		x[i] = y[i] = 1;
	}
	const int size = N * sizeof(float);
	hipMalloc( (void**)&d_x, size); 
	hipMalloc( (void**)&d_y, size); 
	GPUDevice g0 = GPUDevice(0,"saxpy",numBlocks,numThreadsPerBlock);
	g0.startReading();
	for (int i = 0; i < 1000; i++) {
		hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
		hipEventRecord(start,0);
		saxpy<<<numBlocks,numThreadsPerBlock>>>(N, 2.0, d_x, d_y); //numblocks, numthreadsperblock
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&kerneltime,start,stop);
		hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);
	}
	g0.stopReading();
	hipFree(d_x);
	hipFree(d_y);

	

	return 0;
}














/*

#include <stdio.h>

#include <stdlib.h>
#include <iostream>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

#include "nvmlAPI.hpp"

using namespace std;

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char **argv) {

	printf("Saxpy\n");

	int numBlocks, numThreadsPerBlock, N;

	hipEvent_t start,stop;
	float kerneltime;

	if (argc < 3) {
		numBlocks = 2000;
		numThreadsPerBlock = 512;
		N = numBlocks * numThreadsPerBlock;
	} else {
		numBlocks = atoi(argv[1]);
		numThreadsPerBlock = atoi(argv[2]);
		N = numBlocks * numThreadsPerBlock;
	}

	// cout << "\n" << "No. of blocks: " << numBlocks << "\nThreads per block: " << numThreadsPerBlock << "\nTotal no. of threads = " << N << "\n\n";

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// kernel
	// for (i = 0; i < 1000; i++)
	float *d_x, *d_y;
	float x[N], y[N];
	for (int i = 0; i < N; i++) {
		x[i] = y[i] = 1;
	}
	const int size = N * sizeof(float);
	hipMalloc( (void**)&d_x, size); 
	hipMalloc( (void**)&d_y, size); 
	

	nvmlAPIRun();



	for (int i = 0; i < 1000; i++) {

		hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
		hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

		hipEventRecord(start,0);

		saxpy<<<numBlocks,numThreadsPerBlock>>>(N, 2.0, d_x, d_y); //numblocks, numthreadsperblock

		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&kerneltime,start,stop);

		// cout << "Execution time: " << kerneltime << " ms\n";

		hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);

	}

	nvmlAPIEnd();

	
	hipFree(d_x);
	hipFree(d_y);

	

	return 0;
}



*/
